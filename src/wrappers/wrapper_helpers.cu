#include "hip/hip_runtime.h"

#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/reduction.cuh"
#include "hip/hip_runtime.h"

static cuda_time *_total;
static cuda_time *_kernel;
static cuda_time *_reduce;
static cuda_time *_setupcopy;
static unsigned char *pinned_text;

__device__ int warmup_memory = 0;
__global__ void warmup_kernel(){ warmup_memory ^= 1; }

__constant__ unsigned char cpattern[32768];

void wrapper_setup(
	search_parameters p, unsigned char **d_text, unsigned char **d_pattern, int **d_match)
{
	wrapper_setup_malloc(p, d_text, d_pattern, d_match);
    // Warmup kernel to filter out startup overhead
    warmup_kernel<<<1, 1>>>();

	gpuErrchk( hipEventRecord(_total->start) );
	gpuErrchk( hipEventRecord(_setupcopy->start) );
	wrapper_setup_memcpy(p, d_text, d_pattern, d_match);
	gpuErrchk( hipEventRecord(_setupcopy->stop) );
}

void wrapper_setup_malloc(
	search_parameters p, unsigned char **d_text, unsigned char **d_pattern, int **d_match)
{
	size_t text_alloc_size = (p.text_size+p.pattern_size+1) * sizeof(unsigned char);

	gpuErrchk( hipMalloc((void**)d_text, 	 text_alloc_size));
	gpuErrchk( hipMalloc((void**)d_match,   p.text_size * sizeof(int)) );
	if (!p.constant_memory)
		gpuErrchk( hipMalloc((void**)d_pattern, p.pattern_size * sizeof(unsigned char)) );

	if (p.pinned_memory) {
		gpuErrchk( hipHostMalloc((void**)&pinned_text, text_alloc_size) );
		memcpy(pinned_text, p.text, p.text_size * sizeof(unsigned char));
	}
}

void wrapper_setup_memcpy(
	search_parameters p, unsigned char **d_text, unsigned char **d_pattern, int **d_match)
{
	unsigned char* text_ptr;
	if (p.pinned_memory)
		text_ptr = pinned_text;
	else
		text_ptr = p.text;

	gpuErrchk( hipMemcpy(*d_text, text_ptr, 
		p.text_size * sizeof(unsigned char), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemset(*d_match, 0, p.text_size * sizeof(int)) );
	if (p.constant_memory) {
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cpattern), p.pattern, p.pattern_size * sizeof(unsigned char), 0, hipMemcpyHostToDevice) );
		gpuErrchk( hipGetSymbolAddress((void**)d_pattern, cpattern) );
	} else {
		gpuErrchk( hipMemcpy(*d_pattern, p.pattern, 
		p.pattern_size * sizeof(unsigned char), hipMemcpyHostToDevice) );
	}	
}

void wrapper_teardown(
	search_parameters p, search_info *timers, 
	unsigned char *d_text, unsigned char *d_pattern, int *d_match)
{
	int* d_match_count;
	if (p.gpu_reduction){
		gpuErrchk( hipMalloc((void**)&d_match_count, sizeof(int)) );
		gpuErrchk( hipEventRecord(_reduce->start) );
		device_reduce_block_atomic(d_match, d_match_count, p.text_size);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipMemcpy(p.match, d_match_count, sizeof(int), hipMemcpyDeviceToHost) );
		gpuErrchk( hipEventRecord(_reduce->stop) );
	}
	else {
		gpuErrchk( hipMemcpy(p.match, d_match, 
			p.text_size * sizeof(int), hipMemcpyDeviceToHost) );
	}

	gpuErrchk( hipEventRecord(_total->stop) );

	gpuErrchk( hipFree(d_text) );
	gpuErrchk( hipFree(d_match) );
	if (!p.constant_memory)
		gpuErrchk( hipFree(d_pattern) );
	if (p.gpu_reduction) {
		gpuErrchk( hipFree(d_match_count) );
	}
	if (p.pinned_memory) {
		gpuErrchk( hipHostFree(pinned_text) );
	}

	gpuErrchk( hipEventSynchronize(_total->stop) );
	hipError_t code;
	code = hipEventElapsedTime(
		&(timers->kernel_duration), _kernel->start, _kernel->stop);
	if (code == hipErrorInvalidHandle) timers->kernel_duration = 0;
	code = hipEventElapsedTime(
		&(timers->total_duration), _total->start, _total->stop);
	if (code == hipErrorInvalidHandle) timers->total_duration = 0;
	code = hipEventElapsedTime(
		&(timers->setupcopy_duration), _setupcopy->start, _setupcopy->stop);
	if (code == hipErrorInvalidHandle) timers->setupcopy_duration = 0;
	if (p.gpu_reduction){
		gpuErrchk( hipEventElapsedTime(
			&(timers->reduce_duration), _reduce->start, _reduce->stop) );
	}

	gpuErrchk( hipEventDestroy(_total->start) );
	gpuErrchk( hipEventDestroy(_total->stop) );
	gpuErrchk( hipEventDestroy(_kernel->start) );
	gpuErrchk( hipEventDestroy(_kernel->stop) );
	gpuErrchk( hipEventDestroy(_reduce->start) );
	gpuErrchk( hipEventDestroy(_reduce->stop) );
	gpuErrchk( hipEventDestroy(_setupcopy->start) );
	gpuErrchk( hipEventDestroy(_setupcopy->stop) );

	free(_reduce);
	free(_setupcopy);
}

void get_kernel_configuration(
	search_parameters p, unsigned int *grid_dim, unsigned int *block_dim)
{
	get_kernel_configuration_shared(p, 0, grid_dim, block_dim);
}

void get_kernel_configuration_shared(
	search_parameters p, int shared_size,
	unsigned int *grid_dim, unsigned int *block_dim)
{
	int block_count = divUp(p.text_size, p.stride_length);

	if (shared_size != 0){
		int block_dim_max = 
			((shared_size - p.pattern_size) / p.stride_length / 32) * 32;
		if (block_dim_max > block_count)
			*block_dim = min( (int)divUp(block_count, 32) * 32, p.block_dim );
		else
			*block_dim = min( block_dim_max, p.block_dim );
	}
	else
		*block_dim = block_count > p.block_dim ?
		 p.block_dim : divUp(block_count, 32) * 32;


	*grid_dim = divUp(p.text_size, (*block_dim) * p.stride_length);
}

void setup_timers(cuda_time *kernel, cuda_time *total)
{
	_total = total;
	_kernel = kernel;
	_reduce = (struct cuda_time*)malloc(sizeof(struct cuda_time));
	_setupcopy = (struct cuda_time*)malloc(sizeof(struct cuda_time));
	gpuErrchk( hipEventCreate(&(_total->start)) );
	gpuErrchk( hipEventCreate(&(_total->stop)) );
	gpuErrchk( hipEventCreate(&(_kernel->start)) );
	gpuErrchk( hipEventCreate(&(_kernel->stop)) );
	gpuErrchk( hipEventCreate(&(_reduce->start)) );
	gpuErrchk( hipEventCreate(&(_reduce->stop)) );
	gpuErrchk( hipEventCreate(&(_setupcopy->start)) );
	gpuErrchk( hipEventCreate(&(_setupcopy->stop)) );
}
