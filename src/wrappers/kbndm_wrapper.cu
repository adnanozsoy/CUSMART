#include "kbndm_wrapper.h"
#include "algos/kbndm.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"
#include "stddef.h"

search_info factorized_backward_nondeterministic_dawg_matching_wrapper(search_parameters params){

        cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;

	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);
	
        unsigned int d_M;
        unsigned int *d_L, **d_B;
        unsigned int **d_B2 = (unsigned int **)malloc(SIGMA * sizeof(unsigned int *));
        gpuErrchk( hipMalloc((void***)&d_B, SIGMA * sizeof(unsigned int *)) );
        gpuErrchk( hipMalloc((void**)&d_L, SIGMA * sizeof(unsigned int)) );
	
	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);

	/* Preprocessing */
	TicTocTimer preprocess_timer_start = tic();
        int i, k, m1;
        int beg, end;
        unsigned int h_B[SIGMA][SIGMA] = {{0}};
        unsigned int L[SIGMA] = {0};
	/* Preprocessing */
	end = 0;
	for (k = 0; k < (sizeof(unsigned int)*8)-1; k++) {
	  char occ[SIGMA] = {0};
	  while (end < params.pattern_size && occ[params.pattern[end]] == 0) {
	    occ[params.pattern[end]] = 1;
	    end++;
	  }
	}
	m1 = end;
	end = k = 0;
	beg = 0;
	for (;;) {
	  char occ[SIGMA] = {0};
	  while (end < m1 && occ[params.pattern[m1-1-end]] == 0) {
	    occ[params.pattern[m1-1-end]] = 1;
	    end++;
	  }
	  for (i = beg+1; i < end; i++)
	    h_B[params.pattern[m1-i]][params.pattern[m1-1-i]] |= 1L << k;
	  if (end < m1) {
	    h_B[params.pattern[m1-end]][params.pattern[m1-1-end]] |= 1L << k;
	    L[params.pattern[m1-end]] |= 1L << k;
	  } else {
	    d_M = 1L << k;
	    if (end > beg+1) {
	      L[params.pattern[m1-end+1]] |= 1L << k;
	      d_M <<= 1;
	    }
	    break;
	  }
	  beg = end;
	  k++;
	}
	double preprocess_duration = toc(&preprocess_timer_start) * 1000;
        for(i = 0; i < SIGMA; i++){
          gpuErrchk( hipMalloc((void**) &(d_B2[i]), SIGMA*sizeof(unsigned int)) );
          gpuErrchk( hipMemcpy(d_B2[i], h_B[i], SIGMA*sizeof(unsigned int), hipMemcpyHostToDevice) );
        }

        hipMemcpy(d_L, L, SIGMA * sizeof(unsigned int), hipMemcpyHostToDevice);
        gpuErrchk( hipMemcpy(d_B, d_B2, SIGMA*sizeof(float *), hipMemcpyHostToDevice) );
	
	//Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
        factorized_backward_nondeterministic_dawg<<<grid_dim, block_dim>>>(
                                      d_text, params.text_size, d_pattern, params.pattern_size,
                                      m1, d_M, d_B, d_L, params.stride_length, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );
	
	search_info timers = {0};
	timers.preprocess_duration = preprocess_duration;
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	// Release memory
        gpuErrchk( hipFree(d_L) );
        gpuErrchk( hipFree(d_B) );
        for(i = 0; i < SIGMA; i++){
          gpuErrchk( hipFree(d_B2[i]) );
        }
	
	return timers;
	
}
