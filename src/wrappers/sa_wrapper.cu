#include "sa_wrapper.h"
#include "algos/sa.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"
#include <stdio.h>
#include <stdlib.h>

search_info sa_wrapper(search_parameters params){
  
        cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;
	
	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);
	
	unsigned int *d_S, d_D, d_F;
	gpuErrchk( hipMalloc(&d_S, SIGMA * sizeof(unsigned int)) );
	
	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);
	
	/* Preprocessing */
	TicTocTimer preprocess_timer_start = tic();
	unsigned int *h_S;
	h_S = (unsigned int *)malloc(SIGMA * sizeof(unsigned int));   
	
	preSA(params.pattern, params.pattern_size, h_S); 
	d_D = 0;
	d_F = 1<<(params.pattern_size - 1);
	double preprocess_duration = toc(&preprocess_timer_start) * 1000;
	
	gpuErrchk( hipMemcpy(d_S, h_S, SIGMA * sizeof(unsigned int), hipMemcpyHostToDevice) );

	// Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	if(params.pattern_size > WORD){
	      shift_and_large<<<grid_dim, block_dim>>>(d_text, params.text_size, d_pattern,
						       params.pattern_size, d_S, d_D, d_F, params.stride_length, d_match);
	}
	else{
	      shift_and<<<grid_dim, block_dim>>>(d_text, params.text_size, d_pattern,
						params.pattern_size, d_S, d_D, d_F, params.stride_length, d_match);
	}
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );
	
	search_info timers = {0};
	timers.preprocess_duration = preprocess_duration;
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	
	// Release memory
	gpuErrchk( hipFree(d_S) );
	free(h_S); 
	
	return timers;
}
