
#include "bfb_wrapper.h"
#include "algos/bfb.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"

#include <stdio.h>

search_info brute_force_block_wrapper(search_parameters params){

	cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;

	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);

	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);

	// Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	brute_force_block<<<grid_dim, block_dim>>>(
		d_text, params.text_size, d_pattern, params.pattern_size, 
		params.stride_length, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );

	search_info timers = {0};
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	return timers;
}
