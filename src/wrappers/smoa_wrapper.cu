
#include "smoa_wrapper.h"
#include "algos/smoa.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"


search_info string_matching_ordered_alphabet_wrapper(search_parameters params){
	
	cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;
	
	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);

	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);
	

	//Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	
 
	string_matching_ordered_alphabet<<<grid_dim, block_dim>>>(
			d_text, params.text_size, d_pattern, params.pattern_size, 
			params.stride_length, d_match);		
		
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );

	search_info timers = {0};
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);

	return timers;
}
