
#include "ildm2_wrapper.h"
#include "algos/ildm2.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"

search_info improved_linear_dawg2_wrapper(search_parameters params)
{

    cuda_time kernel_time = {0}, total_time = {0};
    unsigned char *d_text, *d_pattern;
    int *d_match;
    unsigned int grid_dim, block_dim;

    setup_timers(&kernel_time, &total_time);
    get_kernel_configuration(params, &grid_dim, &block_dim);

    int *d_ttrans;
    int *d_ttransSMA;
    unsigned char *d_tterminal;

    gpuErrchk( hipMalloc(&d_ttrans, 3*params.pattern_size*SIGMA*sizeof(int)) );
    gpuErrchk( hipMalloc(&d_ttransSMA,
                          (params.pattern_size+1)*SIGMA*sizeof(int)) );
    gpuErrchk( hipMalloc(&d_tterminal, 3*params.pattern_size*sizeof(char)) );

    // Setup: malloc > timer > memset/memcpy
    wrapper_setup(params, &d_text, &d_pattern, &d_match);

    /* Preprocessing */
    TicTocTimer preprocess_timer_start = tic();
    int *h_ttrans, *h_tlength, *h_tsuffix, *h_ttransSMA;
    unsigned char *h_tterminal;

    h_ttrans = (int *)malloc(3*params.pattern_size*SIGMA*sizeof(int));
    memset(h_ttrans, -1, 3*params.pattern_size*SIGMA*sizeof(int));
    h_tlength = (int *)calloc(3*params.pattern_size, sizeof(int));
    h_tsuffix = (int *)calloc(3*params.pattern_size, sizeof(int));
    h_tterminal = (unsigned char *)calloc(3*params.pattern_size, sizeof(char));

    unsigned char *xR = (unsigned char*) malloc (sizeof(char)*(params.pattern_size+1));
    for (int i=0; i<params.pattern_size;
         i++) xR[i] = params.pattern[params.pattern_size-i-1];
    xR[params.pattern_size] = '\0';

    buildSimpleSuffixAutomaton(xR, params.pattern_size, h_ttrans, h_tlength,
                               h_tsuffix, h_tterminal);

    h_ttransSMA = (int *)malloc((params.pattern_size+1)*SIGMA*sizeof(int));
    memset(h_ttransSMA, -1, (params.pattern_size+1)*SIGMA*sizeof(int));
    preSMA(params.pattern, params.pattern_size, h_ttransSMA);
    double preprocess_duration = toc(&preprocess_timer_start) * 1000;
    
    gpuErrchk( hipMemcpy(d_ttrans, h_ttrans,
                          3*params.pattern_size*SIGMA*sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_ttransSMA, h_ttransSMA,
                          (params.pattern_size+1)*SIGMA*sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_tterminal, h_tterminal,
                          3*params.pattern_size*sizeof(char), hipMemcpyHostToDevice) );

    // Kernel run
    gpuErrchk( hipEventRecord(kernel_time.start) );
    improved_linear_dawg2<<<grid_dim, block_dim>>>(
        d_text, params.text_size, d_pattern, params.pattern_size,
        d_ttrans, d_ttransSMA, d_tterminal, params.stride_length, d_match);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipEventRecord(kernel_time.stop) );

    search_info timers = {0};
    timers.preprocess_duration = preprocess_duration;
    // Teardown: copy match back > timer stop > free
    wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
    // Release memory
    gpuErrchk( hipFree(d_ttrans) );
    gpuErrchk( hipFree(d_ttransSMA) );
    gpuErrchk( hipFree(d_tterminal) );
    free(h_ttrans);
    free(h_tlength);
    free(h_tsuffix);
    free(h_tterminal);
    free(h_ttransSMA);

    return timers;
}
