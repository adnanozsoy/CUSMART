
#include "lbndm_wrapper.h"
#include "algos/lbndm.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"

search_info long_backward_nondeterministic_dawg_wrapper(search_parameters params){

    cuda_time kernel_time = {0}, total_time = {0};
    unsigned char *d_text, *d_pattern;
    int *d_match;
    unsigned int grid_dim, block_dim;

    setup_timers(&kernel_time, &total_time);
    get_kernel_configuration(params, &grid_dim, &block_dim);

    int *d_B;
    hipMalloc((void**)&d_B, SIGMA * sizeof(int));
    int *h_B = (int*)calloc(SIGMA, sizeof(int));

    // Setup: malloc > timer start > memset/memcpy
    wrapper_setup(params, &d_text, &d_pattern, &d_match);

    /* Preprocessing */
    TicTocTimer preprocess_timer_start = tic();
    int k = (params.pattern_size-1)/32+1;
    int m1 = params.pattern_size-1;
    int m2 = m1-k;
    int rmd = params.pattern_size-(params.pattern_size/k)*k;
    for (int i=params.pattern_size/k, l=params.pattern_size; i>0; i--, l-=k)
        for (int j=k; j>0; j--)
            h_B[params.pattern[l-j]] |= 1 << (32-i);
    double preprocess_duration = toc(&preprocess_timer_start) * 1000;
    
    hipMemcpy(d_B, h_B, SIGMA * sizeof(int), hipMemcpyHostToDevice);

    // Kernel run
    gpuErrchk( hipEventRecord(kernel_time.start) );

    long_backward_nondeterministic_dawg<<<grid_dim, block_dim>>>(
        d_text, params.text_size, d_pattern, params.pattern_size, 
        d_B, k, m1, m2, rmd, params.stride_length, d_match);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipEventRecord(kernel_time.stop) );

    search_info timers = {0};
    timers.preprocess_duration = preprocess_duration;
    // Teardown: copy match back > timer stop > free
    wrapper_teardown(params, &timers, d_text, d_pattern, d_match);

    //Release Memory
    hipFree(d_B);
    free(h_B);
    return timers;
}
