
#include "tbm_wrapper.h"
#include "algos/bm.cuh"
#include "algos/tbm.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"
#include <stdlib.h>

search_info turbo_boyer_moore_wrapper(search_parameters params){
	
	cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;
	
	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);

	int *d_bmGs, *d_bmBc;
	gpuErrchk( hipMalloc(&d_bmGs, params.pattern_size * sizeof(int)) );
	gpuErrchk( hipMalloc(&d_bmBc, SIGMA * sizeof(int)) );

	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);
	

	/* Preprocessing */
	TicTocTimer preprocess_timer_start = tic();
	int h_bmBc[SIGMA]; 
	int *h_bmGs = (int*)malloc(params.pattern_size * sizeof(int));
	preBmGs(params.pattern, params.pattern_size, h_bmGs);
	preBmBc(params.pattern, params.pattern_size, h_bmBc);
	double preprocess_duration = toc(&preprocess_timer_start) * 1000;
	
	gpuErrchk( hipMemcpy(d_bmGs, h_bmGs, params.pattern_size * sizeof(int), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_bmBc, h_bmBc, SIGMA * sizeof(int), hipMemcpyHostToDevice) );

	//Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	turbo_boyer_moore<<<grid_dim, block_dim>>>(
		d_text, params.text_size, d_pattern, params.pattern_size, 
		d_bmGs, d_bmBc, params.stride_length, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );

	search_info timers = {0};
	timers.preprocess_duration = preprocess_duration;
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	// Release memory
	gpuErrchk( hipFree(d_bmGs) );
	gpuErrchk( hipFree(d_bmBc) );
	free(h_bmGs);

	return timers;
}
