
#include "bmh_sbndm_wrapper.h"
#include "algos/bmh_sbndm.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"

#include <stdlib.h>
#include <string.h>

search_info horspool_with_bndm_wrapper(
    search_parameters params)
{

    cuda_time kernel_time = {0}, total_time = {0};
    unsigned char *d_text, *d_pattern;
    int *d_match;
    unsigned int grid_dim, block_dim;

    setup_timers(&kernel_time, &total_time);
    get_kernel_configuration(params, &grid_dim, &block_dim);

    unsigned int* h_B = (unsigned int*) calloc(SIGMA, sizeof(unsigned int));
    unsigned int* d_B;
    hipMalloc((void**)&d_B, SIGMA * sizeof(unsigned int));
    int* h_hbc = (int*) malloc(SIGMA * sizeof(int));
    int* d_hbc;
    hipMalloc((void**)&d_hbc, SIGMA * sizeof(int));

    // Setup: malloc > timer start > memset/memcpy
    wrapper_setup(params, &d_text, &d_pattern, &d_match);

    int p_len = params.pattern_size <= 32 ? params.pattern_size : 32;
    /* Preprocessing */
    TicTocTimer preprocess_timer_start = tic();
    for (int i=0; i<p_len; i++)
        h_B[params.pattern[p_len-i-1]] |= (unsigned int)1<<(i+32-p_len);
    for (int i=0; i<SIGMA; i++) h_hbc[i]=p_len;
    for (int i=0; i<p_len; i++) h_hbc[params.pattern[i]]=p_len-i-1;
    unsigned int D = h_B[params.pattern[p_len-1]];
    int j=1;
    int shift=1;
    for (int i=p_len-2; i>0; i--, j++) {
        if (D & (1<<(p_len-1))) shift = j;
        D = (D<<1) & h_B[params.pattern[i]];
    }
    double preprocess_duration = toc(&preprocess_timer_start) * 1000;
    
    hipMemcpy((d_text + params.text_size), params.pattern,
               params.pattern_size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, SIGMA * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_hbc, h_hbc, params.pattern_size * sizeof(int),
               hipMemcpyHostToDevice);

    // Kernel run
    gpuErrchk( hipEventRecord(kernel_time.start) );
    if (p_len <= 32) {
        horspool_with_bndm<<<grid_dim, block_dim>>>(
            d_text, params.text_size, d_pattern, params.pattern_size,
            d_B, d_hbc, shift, params.stride_length, d_match);
    }
    else {
        horspool_with_bndm_large<<<grid_dim, block_dim>>>(
            d_text, params.text_size, d_pattern, params.pattern_size,
            d_B, d_hbc, shift, params.stride_length, d_match);

    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipEventRecord(kernel_time.stop) );

    search_info timers = {0};
    timers.preprocess_duration = preprocess_duration;
    // Teardown: copy match back > timer stop > free
    wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
    // Free memory
    hipFree(d_B);
    hipFree(d_hbc);
    free(h_B);
    free(h_hbc);
    return timers;
}
