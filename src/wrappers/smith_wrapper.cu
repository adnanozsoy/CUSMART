
#include "smith_wrapper.h"
#include "algos/smith.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"

#include <stdlib.h>

search_info smith_wrapper(search_parameters params){
  
        cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;
	
	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);
	
	unsigned char *d_qsBc, *d_bmBc;
	gpuErrchk( hipMalloc(&d_qsBc, SIGMA * sizeof(unsigned char)) );
	gpuErrchk( hipMalloc(&d_bmBc, SIGMA * sizeof(unsigned char)) );
	
	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);
	
	/* Preprocessing */
	TicTocTimer preprocess_timer_start = tic();
	unsigned char *h_bmBc, *h_qsBc;
	int malloc_size = SIGMA * sizeof(unsigned char);
	h_bmBc = (unsigned char *)malloc(malloc_size);
	h_qsBc = (unsigned char *)malloc(malloc_size);   
	
	preBmBcSMITH(params.pattern, params.pattern_size, h_bmBc);
	preQsBcSMITH(params.pattern, params.pattern_size, h_qsBc);
	double preprocess_duration = toc(&preprocess_timer_start) * 1000;
	
	gpuErrchk( hipMemcpy(d_qsBc, h_qsBc, SIGMA * sizeof(unsigned char), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_bmBc, h_bmBc, SIGMA * sizeof(unsigned char), hipMemcpyHostToDevice) );
	
	// Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	smith<<<grid_dim, block_dim>>>(d_text, params.text_size, d_pattern,
				       params.pattern_size, d_bmBc, d_qsBc, params.stride_length, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );
	
	search_info timers = {0};
	timers.preprocess_duration = preprocess_duration;
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	
	// Release memory
	gpuErrchk( hipFree(d_bmBc) );
	gpuErrchk( hipFree(d_qsBc) ); 
	free(h_bmBc);
	free(h_qsBc); 
	
	return timers;
}
