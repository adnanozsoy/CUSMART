#include "sfbom_wrapper.h"
#include "algos/sfbom.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "stddef.h"
#include "util/tictoc.h"

search_info sfbom_wrapper(search_parameters params)
{
    cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;

	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);
	
	int **d_trans;
	int **d_trans2 = (int **)malloc((params.pattern_size+2) * sizeof(int *));
	gpuErrchk( hipMalloc((void***)&d_trans, (params.pattern_size+2) * sizeof(int *)) );
	int *d_lambda;
	gpuErrchk( hipMalloc((void**)&d_lambda, (SIGMA*SIGMA)*sizeof(int)) );
	
	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);

	/* Preprocessing */
	TicTocTimer preprocess_timer_start = tic();
	int *S = (int *)malloc((params.pattern_size+1) * sizeof(int));
	int **h_trans = (int **)malloc((params.pattern_size+2) * sizeof(int *));;
	int i, j, p, q, iMinus1, c;
	int LAMBDA[SIGMA*SIGMA];
	
	for (i=0; i<=params.pattern_size+1; i++) h_trans[i] = (int *)malloc (sizeof(int)*(SIGMA)); 
	for (i=0; i<=params.pattern_size+1; i++) for (j=0; j<SIGMA; j++) h_trans[i][j]=UNDEFINED; 
	S[params.pattern_size] = params.pattern_size + 1; 
	for (i = params.pattern_size; i > 0; --i) { 
	  iMinus1 = i - 1; 
	  c = params.pattern[iMinus1]; 
	  h_trans[i][c] = iMinus1; 
	  p = S[i]; 
	  while (p <= params.pattern_size && (q = h_trans[p][c]) ==  UNDEFINED) { 
	    h_trans[p][c] = iMinus1; 
	    p = S[p]; 
	  } 
	  S[iMinus1] = (p == params.pattern_size + 1 ? params.pattern_size : q); 
	} 

	/* Construct the First transition table */ 
	for (i=0; i<SIGMA; i++) { 
	  q = h_trans[params.pattern_size][i]; 
	  for (j=0; j<SIGMA; j++) 
	    if (q>=0) { 
	      if ((p=h_trans[q][j])>=0) FT(i,j) = p; 
	      else FT(i,j)=params.pattern_size+params.pattern_size+1; 
	    } 
	    else FT(i,j) = params.pattern_size+params.pattern_size+1; 
	} 
	q = h_trans[params.pattern_size][params.pattern[params.pattern_size-1]]; 
	for (i=0; i<SIGMA; i++) FT(i,params.pattern[params.pattern_size-1]) = q; 
	for (i=0; i<SIGMA; i++) if (FT(params.pattern[0],i)>params.pattern_size) FT(params.pattern[0],i)-=1; 
	double preprocess_duration = toc(&preprocess_timer_start) * 1000;
	
	hipMemcpy((d_text + params.text_size), params.pattern,
		   params.pattern_size * sizeof(char), hipMemcpyHostToDevice);
	
	for(i = 0; i < (params.pattern_size+2); i++){
          gpuErrchk( hipMalloc((void**) &(d_trans2[i]), (SIGMA)*sizeof(int)) );
          gpuErrchk( hipMemcpy(d_trans2[i], h_trans[i], (SIGMA)*sizeof(int), hipMemcpyHostToDevice) );
        }

	gpuErrchk( hipMemcpy(d_lambda, LAMBDA, (SIGMA*SIGMA)*sizeof(int), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_trans, d_trans2, (params.pattern_size+2)*sizeof(int *), hipMemcpyHostToDevice) );
	
	//Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	sfbom<<<grid_dim, block_dim>>>(
				       d_text, params.text_size, d_pattern, params.pattern_size,
				       d_lambda, d_trans, params.stride_length, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );
	
	search_info timers = {0};
	timers.preprocess_duration = preprocess_duration;
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	// Release memory
	gpuErrchk( hipFree(d_trans) );
	gpuErrchk( hipFree(d_lambda) );
	for(i = 0; i < (params.pattern_size+1); i++){
          gpuErrchk( hipFree(d_trans2[i]) );
        }
	free(S);
	for (i=0; i<params.pattern_size+1; i++) free(h_trans[i]);
	return timers;
}
