
#include "bf_wrapper.h"
#include "algos/bf.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"

#include <stdio.h>

search_info brute_force_wrapper(search_parameters params){

	cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;
	params.stride_length = 1;

	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);

	// Setup: malloc > timer start > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);

	// Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	brute_force<<<grid_dim, block_dim>>>(d_text, params.text_size, d_pattern, params.pattern_size, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );

	search_info timers = {0};
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	return timers;
}
