
#include "trf_wrapper.h"
#include "algos/trf.cuh"
#include "wrapper_helpers.h"
#include "util/cutil.cuh"
#include "util/tictoc.h"

search_info turbo_reverse_factor_wrapper(search_parameters params){

	cuda_time kernel_time = {0}, total_time = {0};
	unsigned char *d_text, *d_pattern;
	int *d_match;
	unsigned int grid_dim, block_dim;
	
	setup_timers(&kernel_time, &total_time);
	get_kernel_configuration(params, &grid_dim, &block_dim);
	
        int *d_ttrans, *d_tshift, *d_mpNext;
        unsigned char *d_tterminal;
	int size = 2 * params.pattern_size + 3;
        int ttrans_size = size*SIGMA*sizeof(int);
	gpuErrchk( hipMalloc(&d_ttrans, ttrans_size) );
	gpuErrchk( hipMalloc(&d_tterminal, size*sizeof(char)) );	
	gpuErrchk( hipMalloc(&d_tshift, size*SIGMA*sizeof(int)) );
	gpuErrchk( hipMalloc(&d_mpNext, (params.pattern_size+1)*sizeof(int)) );
	
	// Setup: malloc > timer > memset/memcpy
	wrapper_setup(params, &d_text, &d_pattern, &d_match);

	/* Preprocessing */
	TicTocTimer preprocess_timer_start = tic();
	int  *ttrans, *tlength, *tposition, *tsuffix, *tshift, *mpNext;
	unsigned char *tterminal;
	
	mpNext = (int *)malloc((params.pattern_size+1)*sizeof(int));
	ttrans = (int *)malloc(ttrans_size);
	tshift = (int *)malloc(size*SIGMA*sizeof(int));
	tlength = (int *)calloc(size, sizeof(int));
	tposition = (int *)calloc(size, sizeof(int));
	tsuffix = (int *)calloc(size, sizeof(int));
	tterminal = (unsigned char *)calloc(size, sizeof(char));
	memset(ttrans, -1, ttrans_size);
	buildSuffixAutomaton4TRF(params.pattern, params.pattern_size, ttrans, tlength, tposition, tsuffix, tterminal, tshift);
	preMpforTRF(params.pattern, params.pattern_size, mpNext);
	double preprocess_duration = toc(&preprocess_timer_start) * 1000;
	
	gpuErrchk( hipMemcpy(d_ttrans, ttrans, 
			      ttrans_size, hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_tterminal, tterminal, 
			      size*sizeof(char), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_tshift, tshift,
			      size*SIGMA*sizeof(int), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_mpNext, mpNext,
			      (params.pattern_size+1)*sizeof(int), hipMemcpyHostToDevice) );
	// Kernel run
	gpuErrchk( hipEventRecord(kernel_time.start) );
	turbo_reverse_factor<<<grid_dim, block_dim>>>(
						d_text, params.text_size, d_pattern, params.pattern_size, 
						d_ttrans, d_tterminal, d_tshift, d_mpNext, params.stride_length, d_match);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipEventRecord(kernel_time.stop) );

	search_info timers = {0};
	timers.preprocess_duration = preprocess_duration;
	// Teardown: copy match back > timer stop > free
	wrapper_teardown(params, &timers, d_text, d_pattern, d_match);
	// Release memory
	gpuErrchk( hipFree(d_ttrans) );
	gpuErrchk( hipFree(d_tterminal) );
	gpuErrchk( hipFree(d_tshift) );
	gpuErrchk( hipFree(d_mpNext) );
	free(mpNext);
	free(tshift);
	free(ttrans);
	free(tlength);
	free(tsuffix);
	free(tterminal);
	
	return timers;
}
