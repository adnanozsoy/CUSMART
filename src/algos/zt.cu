#include "hip/hip_runtime.h"
#include "zt.cuh"

__global__ void zhu_takaoka(unsigned char *text, unsigned long text_size,
			   unsigned char *pattern, int pattern_size, int *bmGs,
			   int **ztBc, int search_len, int *match) {
	
	unsigned long thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long start_inx = thread_id * search_len;

	unsigned long boundary = start_inx + search_len;
	boundary = boundary > text_size ? text_size : boundary;
	int i;
	unsigned long j = start_inx;

	while (j < boundary && j <= text_size - pattern_size) {
	        i = pattern_size - 1;
		while (i >= 0 && pattern[i] == text[i + j]) {
		  --i;
		}
		if (i < 0) {
		      	match[j] = 1;
			j += bmGs[0];
		}
		else{
		        int a = bmGs[i];
			int b = ztBc[text[j + pattern_size - 2]][text[j + pattern_size - 1]];
			j += ((a) > (b) ? (a) : (b));
		}
	}
}

void suffixesZT(unsigned char *x, int m, int *suff) {
  int f, g, i;

  suff[m - 1] = m;
  g = m - 1;
  for (i = m - 2; i >= 0; --i) {
    if (i > g && suff[i + m - 1 - f] < i - g)
      suff[i] = suff[i + m - 1 - f];
    else {
      if (i < g)
	g = i;
      f = i;
      while (g >= 0 && x[g] == x[g + m - 1 - f])
	--g;
      suff[i] = f - g;
    }
  }
}

void preBmGsZT(unsigned char *x, int m, int bmGs[]) {
  int i, j;
  int *suff = (int *)malloc((m+1) * sizeof(int));

  suffixesZT(x, m, suff);

  for (i = 0; i < m; ++i)
    bmGs[i] = m;
  j = 0;
  for (i = m - 1; i >= 0; --i)
    if (suff[i] == i + 1)
      for (; j < m - 1 - i; ++j)
	if (bmGs[j] == m)
	  bmGs[j] = m - 1 - i;
  for (i = 0; i <= m - 2; ++i)
    bmGs[m - 1 - suff[i]] = m - 1 - i;
  free(suff);
}

void preZtBcZT(unsigned char *x, int m, int ztBc[SIGMA][SIGMA]) {
  int i, j;

  for (i = 0; i < SIGMA; ++i)
    for (j = 0; j < SIGMA; ++j)
      ztBc[i][j] = m;
  for (i = 0; i < SIGMA; ++i)
    ztBc[i][x[0]] = m - 1;
  for (i = 1; i < m - 1; ++i)
    ztBc[x[i - 1]][x[i]] = m - 1 - i;
}
